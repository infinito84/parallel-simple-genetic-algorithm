#include "hip/hip_runtime.h"
int adnSize, xSize, ySize, xBits = 0, yBits = 0;
double nDecimals;

void calcSizes(){
    nDecimals = pow(10, DECIMALS);
    xSize = (maxX - minX) * nDecimals;
    ySize = (maxY - minY) * nDecimals;
    double bits = 1, temp;
    while(!xBits || !yBits){
        temp = pow(2,bits);
        if(xSize < temp && !xBits){
            xBits = bits;
        }
        if(ySize < temp && !yBits){
            yBits = bits;
        }
        bits++;
    }
    adnSize = xBits + yBits;
    printf("Bits: adn(%d), x(%d), y(%d)\n", adnSize, xBits, yBits);
}

__device__ void generateAdn(hiprandState_t *state, Cromosome *temp, int adnSize, int nDecimals, int yBits){
	temp->x = randomBetween(state, minX, maxX);
    temp->y = randomBetween(state, minY, maxY);
	int x = (int)(nDecimals * (temp->x - minX));
	int y = (int)(nDecimals * (temp->y - minX));
	int yCount = 0;
    for(int i = adnSize - 1; i >= 0; i--){
        if(yCount < yBits){
            temp->adn[i] = (int)y % 2;
            y = y / 2;
            yCount++;
        }
        else{
            temp->adn[i] = (int)x % 2;
            x = x / 2;
        }
    }
}

__device__ __host__ void showCromosome(Cromosome *temp, int adnSize){
    printf("Cromosome: adn(");
    for(int i=0;i<adnSize;i++) printf("%d", temp->adn[i]);
    printf("), x(%.2f), y(%.2f), fitness(%f), ruleta(%f)\n", temp->x, temp->y, temp->fitness, temp->before);
}

__device__ Cromosome randomCromosome(hiprandState_t *state, int adnSize, int nDecimals, int yBits){
    Cromosome temp;
    temp.adn = new int[adnSize];
	generateAdn(state, &temp, adnSize, nDecimals, yBits);
	//showCromosome(&temp, adnSize);
    return temp;
}
